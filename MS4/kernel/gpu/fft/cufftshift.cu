#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// Launch configuration should be as follows:
//  1. blocks of dim3(threads_per_dim, threads_per_dim, 1) size
//       where threads_per_dim = min(N, 16)
//  2. grid of dim3((N+threads_per_dim-1)/threads_per_dim, (N-1)/(threads_per_dim * 2)+1, 1) blocks

template <class T>
static __device__ __inline__
void fftshift_kernel_common(T* data, int shift, int N, int pitch){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x > N-1 || y > (N+1)/2-1) return;

  int x1 = x + shift;
  if (x1 > N-1) x1 -= N;

  int y1 = y + shift;
  if (y1 > N-1) y1 -= N;

  int
      i = x + y * pitch
    , i1 = x1 + y1 * pitch;

  T tmp;
  tmp = data[i];
  data[i] = data[i1];
  data[i1] = tmp;
}

extern "C" {
__global__
void fftshift_kernel_cx(hipDoubleComplex* data, int N, int pitch) {
  fftshift_kernel_common<hipDoubleComplex>(data, N/2, N, pitch);
}

__global__
void ifftshift_kernel_cx(hipDoubleComplex* data, int N, int pitch) {
  fftshift_kernel_common<hipDoubleComplex>(data, N/2 + N%2, N, pitch);
}

__global__
void fftshift_kernel_r(double* data, int N, int pitch) {
  fftshift_kernel_common<double>(data, N/2, N, pitch);
}

__global__
void ifftshift_kernel_r(double* data, int N, int pitch) {
  fftshift_kernel_common<double>(data, N/2 + N%2, N, pitch);
}
}
